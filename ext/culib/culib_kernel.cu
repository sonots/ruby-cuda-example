#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ 
void my_kernel(int *ptr, int val, size_t N)
{
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        ptr[i] = val;
    }
}

extern "C"
void my_kernel_launch()
{
  int N = 8;
  int *a;
  hipMallocManaged((void**)&a, sizeof(int)*N);

  //*a = 11;
  //for (int i = 0; i < N; i++)
  //  printf("%d ", a[i]);
  //printf("\n");
  //cudaMemset(a, 0, N*sizeof(int));
  //for (int i = 0; i < N; i++)
  //  printf("%d ", a[i]);
  //printf("\n");
  //for (int i = 0; i < N; i++)
  //  a[i] += 1;
  //for (int i = 0; i < N; i++)
  //  printf("%d ", a[i]);
  //printf("\n");

  my_kernel<<<2,2>>>(a, 10, N);
  hipDeviceSynchronize();
  for (int i = 0; i < N; i++)
    printf("%d ", a[i]);
  printf("\n");
}

