#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ 
void my_kernel(int *ptr, int val, size_t N)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) ptr[i] = val;
    //for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
    //    ptr[i] = val;
    //}
}

extern "C"
void my_kernel_launch()
{
  int N = 8;
  int *a;
  //cudaMallocManaged((void**)&a, sizeof(int)*N);
  hipMalloc((void**)&a, sizeof(int)*N);

  //*a = 11;
  //for (int i = 0; i < N; i++)
  //  printf("%d ", a[i]);
  //printf("\n");
  //cudaMemset(a, 0, N*sizeof(int));
  //for (int i = 0; i < N; i++)
  //  printf("%d ", a[i]);
  //printf("\n");
  //for (int i = 0; i < N; i++)
  //  a[i] += 1;
  //for (int i = 0; i < N; i++)
  //  printf("%d ", a[i]);
  //printf("\n");

  my_kernel<<<2,2>>>(a, 10, N);
  printf("%d\n", *a);
  //for (int i = 0; i < N; i++)
  //  printf("%d ", a[i]);
  //printf("\n");
}

